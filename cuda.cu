// cuda_sum.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define NUM_ELEMENTS 100000000
#define THREADS_PER_BLOCK 256

__global__ void sum_reduction(int *input, long long *output, int n) {
    __shared__ long long shared_data[THREADS_PER_BLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Carrega os dados na memória compartilhada
    long long temp = 0;
    if (i < n)
        temp = input[i];
    shared_data[tid] = temp;
    __syncthreads();

    // Redução na memória compartilhada
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && (i + s) < n) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    // Escreve o resultado deste bloco na memória global
    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

int main() {
    int *h_data = (int *) malloc(NUM_ELEMENTS * sizeof(int));
    int *d_data;
    long long *d_partial_sums, *h_partial_sums;
    long long total_sum = 0;

    // Gera números aleatórios
    printf("Gerando números aleatórios...\n");
    srand(time(NULL));
    for (int i = 0; i < NUM_ELEMENTS; i++) {
        h_data[i] = rand() % 100 + 1;
    }

    // Aloca memória no dispositivo
    hipMalloc((void**)&d_data, NUM_ELEMENTS * sizeof(int));

    int blocks = (NUM_ELEMENTS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipMalloc((void**)&d_partial_sums, blocks * sizeof(long long));
    h_partial_sums = (long long*) malloc(blocks * sizeof(long long));

    // Copia os dados para o dispositivo
    hipMemcpy(d_data, h_data, NUM_ELEMENTS * sizeof(int), hipMemcpyHostToDevice);

    // Lança o kernel e mede o tempo
    printf("\nCalculando a soma usando CUDA...\n");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    sum_reduction<<<blocks, THREADS_PER_BLOCK>>>(d_data, d_partial_sums, NUM_ELEMENTS);

    // Copia as somas parciais de volta para o host
    hipMemcpy(h_partial_sums, d_partial_sums, blocks * sizeof(long long), hipMemcpyDeviceToHost);

    // Redução final no host
    for (int i = 0; i < blocks; i++) {
        total_sum += h_partial_sums[i];
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Soma Total: %lld\n", total_sum);
    printf("Tempo levado usando CUDA: %.2f segundos\n", milliseconds / 1000.0);

    // Libera memória
    hipFree(d_data);
    hipFree(d_partial_sums);
    free(h_data);
    free(h_partial_sums);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
